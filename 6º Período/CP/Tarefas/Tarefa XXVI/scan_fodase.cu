
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void scan_cuda(double* a, double *s, int width) {
  int t = threadIdx.x;
  int b = blockIdx.x * blockDim.x;
  double fodase;

  __shared__ double p[1024];
	
  if(b+t < width) {
     p[t] = a[t+b];
  }

  __syncthreads();

  for(int i = 1; i < blockDim.x; i = i * 2) {
    if (t >= i){
	  fodase = p[t] + p[t-1];
	}

	__syncthreads();

	if (t >= i){
      p[t] = fodase;
	}

	__syncthreads();
  }

  if (b + t < width){
    a[t+b] = p[t];
  }

  if (t == blockDim.x-1){
    s[blockIdx.x+1] = a[t+b];
  }
} 

__global__ void add_cuda(double *a, double *s, int width) {
    int t = threadIdx.x;
    int b = blockIdx.x * blockDim.x;

    if(b+t < width) {
        a[b+t] += s[blockIdx.x];
    }
}

int main()
{
  int width = 40000000;
  int size = width * sizeof(double);

  int block_size = 1024;
  int num_blocks = (width-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));

  double *a = (double*) malloc (size);
  double *s = (double*) malloc (s_size);

  for(int i = 0; i < width; i++)
    a[i] = i;

  double *d_a, *d_s;

  // alocar vetores "a" e "s" no device
  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_s, s_size);
  // copiar vetor "a" para o device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  // definição do número de blocos e threads (dimGrid e dimBlock)
  dim3 dimGrid(num_blocks,1,1);
  dim3 dimBlock(block_size,1,1);
  // chamada do kernel scan
  scan_cuda<<<dimGrid,dimBlock>>>(d_a, d_s, width);
  // copiar vetor "s" para o host
  hipMemcpy(s, d_s, s_size, hipMemcpyDeviceToHost);
  // scan no host (já implementado)
  s[0] = 0;
  for (int i = 1; i < num_blocks; i++)
    s[i] += s[i-1];

  // copiar vetor "s" para o device
  hipMemcpy(d_s, s, s_size, hipMemcpyHostToDevice);
  // chamada do kernel da soma
  add_cuda<<<dimGrid,dimBlock>>>(d_a, d_s, width);
  // copiar o vetor "a" para o host
  hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
  printf("\na[%d] = %f\n",width-1,a[width-1]);

  hipFree(d_a);
  hipFree(d_s);
}
